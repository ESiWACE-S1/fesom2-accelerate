
#include <hip/hip_runtime.h>
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,__double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

/* Block size X: 32 */
__global__ void fct_ale_b1_horizontal(const int maxLevels, const int * __restrict__ nLevels, const int * __restrict__ nodesPerEdge, const int * __restrict__ elementsPerEdge, const double * __restrict__ fct_adf_h, double * __restrict__ fct_plus, double * __restrict__ fct_minus)
{
int levelBound = 0;
const int nodeOne = (nodesPerEdge[(blockIdx.x * 2)] - 1) * maxLevels;
const int nodeTwo = (nodesPerEdge[(blockIdx.x * 2) + 1] - 1) * maxLevels;

/* Compute the upper bound for the level */
levelBound = elementsPerEdge[(blockIdx.x * 2) + 1] - 1;
if ( levelBound > 0 )
{
    levelBound = max(nLevels[elementsPerEdge[(blockIdx.x * 2)] - 1], nLevels[levelBound]);
}
else
{
    levelBound = max(nLevels[elementsPerEdge[(blockIdx.x * 2)] - 1], 0);
}
/* Compute fct_plus and fct_minus */
for ( int level = threadIdx.x; level < levelBound; level += 32 )
{
    double fct_adf_h_value = 0.0;
    fct_adf_h_value = fct_adf_h[(blockIdx.x * maxLevels) + level];
    atomicAdd(&(fct_plus[nodeOne + level]), fmax(0.0, fct_adf_h_value));
    atomicAdd(&(fct_minus[nodeOne + level]), fmin(0.0, fct_adf_h_value));
    atomicAdd(&(fct_plus[nodeTwo + level]), fmax(0.0, -fct_adf_h_value));
    atomicAdd(&(fct_minus[nodeTwo + level]), fmin(0.0, -fct_adf_h_value));
}
}