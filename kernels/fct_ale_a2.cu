
#include <hip/hip_runtime.h>
/* Block size X: 32 */
__global__ void fct_ale_a2(const int * __restrict__ nLevels, const int * __restrict__ elementNodes, double2 * __restrict__ UV_rhs, const double * __restrict__ fct_ttf_max, const double * __restrict__ fct_ttf_min)
{
const int element_index = (blockIdx.x * 48);
const int element_node0_index = elementNodes[(blockIdx.x * 3)] * 48;
const int element_node1_index = elementNodes[(blockIdx.x * 3) + 1] * 48;
const int element_node2_index = elementNodes[(blockIdx.x * 3) + 2] * 48;
for ( int level = threadIdx.x; level < 48 - 1; level += 32 )
{
if ( level < nLevels[blockIdx.x] )
{
double2 temp = make_double2(0.0, 0.0);
temp.x = fmax(fct_ttf_max[element_node0_index + level], fct_ttf_max[element_node1_index + level]);
temp.x = fmax(temp.x, fct_ttf_max[element_node2_index + level]);
temp.y = fmin(fct_ttf_min[element_node0_index + level], fct_ttf_min[element_node1_index + level]);
temp.y = fmin(temp.y, fct_ttf_min[element_node2_index + level]);
UV_rhs[element_index + level] = temp;
}
else
{
UV_rhs[element_index + level] = make_double2(-1.7976931348623157e+308, 1.7976931348623157e+308);
}
}
}
