
#include <fesom2-accelerate.h>


struct gpuMemory * allocate(void * hostMemory, std::size_t size)
{
    hipError_t status = hipSuccess;
    struct gpuMemory * allocatedMemory = new struct gpuMemory;

    allocatedMemory->host_pointer = hostMemory;
    allocatedMemory->size = size;
    status = hipMalloc(&(allocatedMemory->device_pointer), size);
    if ( !errorHandling(status) )
    {
        delete allocatedMemory;
        return nullptr;
    }
    return allocatedMemory;
}
