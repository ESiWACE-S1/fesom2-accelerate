#include "hip/hip_runtime.h"

#include <fesom2-accelerate.h>
#include <hip/hip_vector_types.h>


// CUDA kernels
extern __global__ void fct_ale_a1(const double * __restrict__ fct_low_order, const double * __restrict__ ttf, const int * __restrict__ nLevels, double * fct_ttf_max, double * fct_ttf_min);
extern __global__ void fct_ale_a2(const int * __restrict__ nLevels, const int * __restrict__ elementNodes, double2 * __restrict__ UV_rhs, const double * __restrict__ fct_ttf_max, const double * __restrict__ fct_ttf_min);

struct gpuMemory * allocate(void * hostMemory, std::size_t size)
{
    hipError_t status = hipSuccess;
    struct gpuMemory * allocatedMemory = new struct gpuMemory;

    allocatedMemory->host_pointer = hostMemory;
    allocatedMemory->size = size;
    status = hipMalloc(&(allocatedMemory->device_pointer), size);
    if ( !errorHandling(status) )
    {
        delete allocatedMemory;
        return nullptr;
    }
    return allocatedMemory;
}

void fct_ale_a1_accelerated(const int nNodes, struct gpuMemory * nLevels_nod2D, struct gpuMemory * fct_ttf_max, struct gpuMemory * fct_ttf_min,  struct gpuMemory * fct_low_order, struct gpuMemory * ttf, bool synchronous, hipStream_t stream)
{
    bool status = true;

    status = transferToDevice(*fct_low_order, synchronous, stream);
    if ( !status )
    {
        return;
    }
    status = transferToDevice(*ttf, synchronous, stream);
    if ( !status )
    {
        return;
    }
    fct_ale_a1<<< dim3(nNodes), dim3(32) >>>(reinterpret_cast<real_type *>(fct_low_order->device_pointer), reinterpret_cast<real_type *>(ttf->device_pointer), reinterpret_cast<int *>(nLevels_nod2D->device_pointer), reinterpret_cast<real_type *>(fct_ttf_max->device_pointer), reinterpret_cast<real_type *>(fct_ttf_min->device_pointer));
    status = transferToHost(*fct_ttf_max, synchronous, stream);
    if ( !status )
    {
        return;
    }
    status = transferToHost(*fct_ttf_min, synchronous, stream);
    if ( !status )
    {
        return;
    }
}

void fct_ale_a2_accelerated(const int nElements, const struct gpuMemory * nLevels_elem, struct gpuMemory * elementNodes, struct gpuMemory * UV_rhs, struct gpuMemory * fct_ttf_max, struct gpuMemory * fct_ttf_min, bool synchronous, hipStream_t stream)
{
    bool status = true;

    status = transferToDevice(*fct_ttf_max, synchronous, stream);
    if ( !status )
    {
        return;
    }
    status = transferToDevice(*fct_ttf_min, synchronous, stream);
    if ( !status )
    {
        return;
    }
    fct_ale_a2<<< dim3(nElements), dim3(32) >>>(reinterpret_cast<int *>(nLevels_elem->device_pointer), reinterpret_cast<int *>(elementNodes->device_pointer), reinterpret_cast<real2_type *>(UV_rhs->device_pointer), reinterpret_cast<real_type *>(fct_ttf_max->device_pointer), reinterpret_cast<real_type *>(fct_ttf_min->device_pointer));
    status = transferToHost(*UV_rhs, synchronous, stream);
    if ( !status )
    {
        return;
    }
}
