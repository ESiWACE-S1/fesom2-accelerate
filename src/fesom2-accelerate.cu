#include "hip/hip_runtime.h"
#include <fesom2-accelerate.h>
#include <hip/hip_vector_types.h>
#include <iostream>

// CUDA kernels
extern __global__ void fct_ale_a1(const int maxLevels, const double * __restrict__ fct_low_order, const double * __restrict__ ttf, const int * __restrict__ nLevels, double * fct_ttf_max, double * fct_ttf_min);
extern __global__ void fct_ale_a2(const int maxLevels, const int * __restrict__ nLevels, const int * __restrict__ elementNodes, double2 * __restrict__ UV_rhs, const double * __restrict__ fct_ttf_max, const double * __restrict__ fct_ttf_min);
extern __global__ void fct_ale_a2b(const int maxLevels, const int * __restrict__ nLevels, const int * __restrict__ elementNodes, double * __restrict__ UV_rhs, const double * __restrict__ fct_ttf_max, const double * __restrict__ fct_ttf_min, double bignumber);
extern __global__ void fct_ale_a3(const int maxLevels, const int maxElements, const int * __restrict__ nLevels, const int * __restrict__ elements_in_node, const int * __restrict__ number_elements_in_node, const double2 * __restrict__ UV_rhs, double * __restrict__ fct_ttf_max, double * __restrict__ fct_ttf_min, const double * __restrict__ fct_lo);
extern __global__ void fct_ale_b1_vertical(const int maxLevels, const int * __restrict__ nLevels, const double * __restrict__ fct_adf_v, double * __restrict__ fct_plus, double * __restrict__ fct_minus);
extern __global__ void fct_ale_b1_horizontal(const int maxLevels, const int * __restrict__ nLevels, const int * __restrict__ nodesPerEdge, const int * __restrict__ elementsPerEdge, const double * __restrict__ fct_adf_h, double * __restrict__ fct_plus, double * __restrict__ fct_minus);
extern __global__ void fct_ale_b2(const int maxLevels, const double dt, const double fluxEpsilon, const int * __restrict__ nLevels, const double * __restrict__ area_inv, const double * __restrict__ fct_ttf_max, const double * __restrict__ fct_ttf_min, double * __restrict__ fct_plus, double * __restrict__ fct_minus);
extern __global__ void fct_ale_pre_comm(const int max_levels, const int num_nodes, const int max_num_elems, const int * __restrict__ node_levels, const int * __restrict__ elem_levels, const int * __restrict__ node_elems, const int * __restrict__ node_num_elems, const int * __restrict__ elem_nodes, const double * __restrict__ fct_low_order, const double * __restrict__ ttf, const double * __restrict__ fct_adf_v, const double * __restrict__ fct_adf_h, double * __restrict__ UVrhs, double * __restrict__ fct_ttf_max, double * __restrict__ fct_ttf_min, double * __restrict__ fct_plus, double * __restrict__ fct_minus, const double bignr);
extern __global__ void fct_ale_b3_vertical(const int maxLevels, const int * __restrict__ nLevels, double * __restrict__ fct_adf_v, const double * __restrict__ fct_plus, const double * __restrict__ fct_minus);
extern __global__ void fct_ale_b3_horizontal(const int maxLevels, const int * __restrict__ nLevels, const int * __restrict__ nodesPerEdge, const int * __restrict__ elementsPerEdge, double * __restrict__ fct_adf_h, const double * __restrict__ fct_plus, const double * __restrict__ fct_minus);

struct gpuMemory * allocate(void * hostMemory, std::size_t size)
{
    hipError_t status = hipSuccess;
    struct gpuMemory * allocatedMemory = new struct gpuMemory;

    allocatedMemory->host_pointer = hostMemory;
    allocatedMemory->size = size;
    status = hipMalloc(&(allocatedMemory->device_pointer), size);
    if ( !errorHandling(status) )
    {
        delete allocatedMemory;
        return nullptr;
    }
    return allocatedMemory;
}

void fct_ale_a1_accelerated(const int maxLevels, const int nNodes, struct gpuMemory * nLevels_nod2D, struct gpuMemory * fct_ttf_max, struct gpuMemory * fct_ttf_min,  struct gpuMemory * fct_low_order, struct gpuMemory * ttf, bool synchronous, hipStream_t stream)
{
    bool status = true;

    status = transferToDevice(*fct_low_order, synchronous, stream);
    if ( !status )
    {
        return;
    }
    status = transferToDevice(*ttf, synchronous, stream);
    if ( !status )
    {
        return;
    }
    fct_ale_a1<<< dim3(nNodes), dim3(32) >>>(maxLevels, reinterpret_cast<real_type *>(fct_low_order->device_pointer), reinterpret_cast<real_type *>(ttf->device_pointer), reinterpret_cast<int *>(nLevels_nod2D->device_pointer), reinterpret_cast<real_type *>(fct_ttf_max->device_pointer), reinterpret_cast<real_type *>(fct_ttf_min->device_pointer));
    status = transferToHost(*fct_ttf_max, synchronous, stream);
    if ( !status )
    {
        return;
    }
    status = transferToHost(*fct_ttf_min, synchronous, stream);
    if ( !status )
    {
        return;
    }
}

void fct_ale_a2_accelerated(const int maxLevels, const int nElements, const struct gpuMemory * nLevels_elem, struct gpuMemory * elementNodes, struct gpuMemory * UV_rhs, struct gpuMemory * fct_ttf_max, struct gpuMemory * fct_ttf_min, bool synchronous, hipStream_t stream)
{
    bool status = true;

    status = transferToDevice(*fct_ttf_max, synchronous, stream);
    if ( !status )
    {
        return;
    }
    status = transferToDevice(*fct_ttf_min, synchronous, stream);
    if ( !status )
    {
        return;
    }
    fct_ale_a2<<< dim3(nElements), dim3(32) >>>(maxLevels, reinterpret_cast<int *>(nLevels_elem->device_pointer), reinterpret_cast<int *>(elementNodes->device_pointer), reinterpret_cast<real2_type *>(UV_rhs->device_pointer), reinterpret_cast<real_type *>(fct_ttf_max->device_pointer), reinterpret_cast<real_type *>(fct_ttf_min->device_pointer));
    status = transferToHost(*UV_rhs, synchronous, stream);
    if ( !status )
    {
        return;
    }
}

void fct_ale_a1_a2_accelerated(const int maxLevels, const int nNodes, const int nElements, struct gpuMemory * nLevels_nod2D, struct gpuMemory * nLevels_elem, struct gpuMemory * elementNodes, struct gpuMemory * fct_ttf_max, struct gpuMemory * fct_ttf_min, struct gpuMemory * fct_low_order, struct gpuMemory * ttf, struct gpuMemory * UV_rhs, bool synchronous, hipStream_t stream)
{
    bool status = true;

    status = transferToDevice(*fct_low_order, synchronous, stream);
    if ( !status )
    {
        return;
    }
    status = transferToDevice(*ttf, synchronous, stream);
    if ( !status )
    {
        return;
    }
    fct_ale_a1<<< dim3(nNodes), dim3(32) >>>(maxLevels, reinterpret_cast<real_type *>(fct_low_order->device_pointer), reinterpret_cast<real_type *>(ttf->device_pointer), reinterpret_cast<int *>(nLevels_nod2D->device_pointer), reinterpret_cast<real_type *>(fct_ttf_max->device_pointer), reinterpret_cast<real_type *>(fct_ttf_min->device_pointer));
    fct_ale_a2<<< dim3(nElements), dim3(32) >>>(maxLevels, reinterpret_cast<int *>(nLevels_elem->device_pointer), reinterpret_cast<int *>(elementNodes->device_pointer), reinterpret_cast<real2_type *>(UV_rhs->device_pointer), reinterpret_cast<real_type *>(fct_ttf_max->device_pointer), reinterpret_cast<real_type *>(fct_ttf_min->device_pointer));
    status = transferToHost(*UV_rhs, synchronous, stream);
    if ( !status )
    {
        return;
    }
}

void transfer_mesh_(void** ret, int* host_ptr, int* size, int* istat)
{
    struct gpuMemory* gpumem = allocate((void*)host_ptr, (*size) * sizeof(int));
    if ( transferToDevice(*gpumem) )
    {
        *ret = (void*)gpumem;
        *istat = 0;
    }
    else
    {
        *ret = nullptr;
        *istat = 1;
    }
}

void alloc_var_(void** ret, real_type* host_ptr, int* size, int* istat)
{
    struct gpuMemory* gpumem = allocate((void*)host_ptr, (*size) * sizeof(real_type));
    *istat = (gpumem == nullptr)?1:0;
    *ret = (void*)gpumem;
}

void allocate_pinned_doubles_(void** hostptr, int* size)
{
    hipError_t status = hipSuccess;
    status = hipHostMalloc(hostptr, sizeof(double) * (*size));
    if ( !errorHandling(status) )
    {
        std::cerr<<"Error in allocating page-locked memory"<<std::endl;
    }
}

void transfer_var_(void** mem, real_type* host_ptr)
{
    struct gpuMemory* mem_gpu = static_cast<gpuMemory*>(*mem);
    mem_gpu->host_pointer = (void*)host_ptr;
    transferToDevice(*mem_gpu, true);
}

void transfer_var_async_(void** mem, real_type* host_ptr)
{
    struct gpuMemory* mem_gpu = static_cast<gpuMemory*>(*mem);
    mem_gpu->host_pointer = (void*)host_ptr;
    transferToDevice(*mem_gpu, false);
}

void reserve_var_(void** ret, int* size, int* istat)
{
    struct gpuMemory* gpumem = allocate(nullptr, (*size) * sizeof(real_type));
    *istat = (gpumem == nullptr)?1:0;
    *ret = (void*)gpumem;
}

std::ostream& operator << (std::ostream& os, const gpuMemory& gpumem)
{
    os<<"host at:"<<gpumem.host_pointer<<", dev at:"<<gpumem.device_pointer<<", size: "<<gpumem.size;
    return os;
}

void set_mpi_rank_(int* rank, int* total_ranks)
{
    int rank_on_node = (*rank) % (*total_ranks);
    int count = 1;
    hipError_t status = hipGetDeviceCount(&count);
    if ( !errorHandling(status) )
    {
        std::cerr<<"Error in getting number of CUDA devices"<<std::endl;
        return;
    }
    if ( count < 1)
    {
        std::cerr<<"No CUDA devices found on node where rank "<<(*rank)<<" runs!"<<std::endl;
        return;
    }
    int device_id = rank_on_node % count;
    status = hipSetDevice(device_id);
    if ( !errorHandling(status) )
    {
        std::cerr<<"Error in setting device id to"<<device_id<<std::endl;
        return;
    }
}

inline void transfer_back(void* memory, const std::string& variable, int* state)
{
    if(*state == 0)
    {
        return;
    }
    bool status =  transferToHost(*static_cast<gpuMemory*>(memory));
    if ( !status )
    {
        std::cerr<<"Error in transfer "<<variable<<" to host"<<std::endl;
        *state = 0;
    }
}

#define NUM_KERNELS 100

void fct_ale_pre_comm_acc_( int* alg_state, void** fct_ttf_max, void**  fct_ttf_min, void**  fct_plus, void**  fct_minus, void** ttf, void** fct_LO, void**  fct_adf_v, void** fct_adf_h, void** UV_rhs, void** area_inv, int* myDim_nod2D, int* eDim_nod2D, int* myDim_elem2D, int* myDim_edge2D, int* nl, void** nlevels_nod2D, void** nlevels_elem2D, void** elem2D_nodes, void** nod_in_elem2D_num, void** nod_in_elem2D, int* nod_in_elem2D_dim, void** nod2D_edges, void** elem2D_edges, int* vlimit, real_type* flux_eps, real_type* bignumber, real_type* dt)
{
#if NUM_KERNELS < 1
    return;
#endif
    *alg_state = 0;
    bool status = true;
    int nNodes = (*myDim_nod2D) + (*eDim_nod2D);

    status = transferToDevice(*static_cast<gpuMemory*>(*fct_LO)); 
    if ( !status )
    {
        std::cerr<<"Error in transfer of fct_LO to device"<<std::endl;
        return;
    }
    // ttf: transferred before fct_ale_muscl_LH
    // fct_adf_v, fct_adf_h: transferred in fct_ale_muscl_LH

    int maxLevels = *nl - 1;
    int maxnElems = *nod_in_elem2D_dim;
    int* nlevels_nod2D_dev = reinterpret_cast<int*>(static_cast<gpuMemory*>(*nlevels_nod2D)->device_pointer);
    int* nlevels_elem2D_dev = reinterpret_cast<int*>(static_cast<gpuMemory*>(*nlevels_elem2D)->device_pointer);
    int* node_elems_dev = reinterpret_cast<int*>(static_cast<gpuMemory*>(*nod_in_elem2D)->device_pointer);
    int* node_num_elems_dev = reinterpret_cast<int*>(static_cast<gpuMemory*>(*nod_in_elem2D_num)->device_pointer);
    int* elem2D_nodes_dev = reinterpret_cast<int*>(static_cast<gpuMemory*>(*elem2D_nodes)->device_pointer);
    int* nod2D_edges_dev = reinterpret_cast<int*>(static_cast<gpuMemory*>(*nod2D_edges)->device_pointer);
    int* elem2D_edges_dev = reinterpret_cast<int*>(static_cast<gpuMemory*>(*elem2D_edges)->device_pointer);
    real_type* fct_lo_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_LO)->device_pointer);
    real_type* ttf_dev    = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*ttf)->device_pointer);
    real_type* fct_adf_v_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_adf_v)->device_pointer);
    real_type* fct_adf_h_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_adf_h)->device_pointer);
    real_type* UV_rhs_dev    = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*UV_rhs)->device_pointer);
    real2_type* UV_rhs_dev2    = reinterpret_cast<real2_type*>(static_cast<gpuMemory*>(*UV_rhs)->device_pointer);
    real_type* fct_ttf_max_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_ttf_max)->device_pointer);
    real_type* fct_ttf_min_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_ttf_min)->device_pointer);
    real_type* fct_plus_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_plus)->device_pointer);
    real_type* fct_min_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_minus)->device_pointer);
    real_type* area_inv_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*area_inv)->device_pointer);

    fct_ale_a1<<< dim3(nNodes), dim3(32) >>>(maxLevels, fct_lo_dev, ttf_dev, nlevels_nod2D_dev, fct_ttf_max_dev, fct_ttf_min_dev);
#if NUM_KERNELS < 2
    *alg_state = 1;
    transfer_back(*fct_ttf_max, "fct_ttf_max", alg_state);
    transfer_back(*fct_ttf_min, "fct_ttf_min", alg_state);
    return;
#endif
    fct_ale_a2<<< dim3(*myDim_elem2D), dim3(32) >>>(maxLevels, nlevels_elem2D_dev, elem2D_nodes_dev, UV_rhs_dev2, fct_ttf_max_dev, fct_ttf_min_dev);
#if NUM_KERNELS < 3
    *alg_state = 2;
    transfer_back(*fct_ttf_max, "fct_ttf_max", alg_state);
    transfer_back(*fct_ttf_min, "fct_ttf_min", alg_state);
    transfer_back(*UV_rhs, "UV_rhs", alg_state);
    return;
#endif
    fct_ale_a3<<< dim3(*myDim_nod2D), dim3(32), 2 * maxLevels * sizeof(real_type) >>>(maxLevels, maxnElems, nlevels_nod2D_dev, node_elems_dev, node_num_elems_dev, UV_rhs_dev2, fct_ttf_max_dev, fct_ttf_min_dev, fct_lo_dev);
#if NUM_KERNELS < 4
    *alg_state = 3;
    transfer_back(*fct_ttf_max, "fct_ttf_max", alg_state);
    transfer_back(*fct_ttf_min, "fct_ttf_min", alg_state);
    return;
#endif
    fct_ale_b1_vertical<<< dim3(*myDim_nod2D), dim3(32) >>>(maxLevels, nlevels_nod2D_dev, fct_adf_v_dev, fct_plus_dev, fct_min_dev);
#if NUM_KERNELS < 5
    *alg_state = 4;
    transfer_back(*fct_ttf_max, "fct_ttf_max", alg_state);
    transfer_back(*fct_ttf_min, "fct_ttf_min", alg_state);
    transfer_back(*fct_plus, "fct_plus", alg_state);
    transfer_back(*fct_minus, "fct_minus", alg_state);
    return;
#endif
    fct_ale_b1_horizontal<<< dim3(*myDim_nod2D), dim3(32) >>>(maxLevels, nlevels_elem2D_dev, nod2D_edges_dev, elem2D_edges_dev, fct_adf_h_dev, fct_plus_dev, fct_min_dev);
#if NUM_KERNELS < 6
    *alg_state = 5;
    transfer_back(*fct_ttf_max, "fct_ttf_max", alg_state);
    transfer_back(*fct_ttf_min, "fct_ttf_min", alg_state);
    transfer_back(*fct_plus, "fct_plus", alg_state);
    transfer_back(*fct_minus, "fct_minus", alg_state);
    return;
#endif
    fct_ale_b2<<< dim3(*myDim_nod2D), dim3(32) >>>(maxLevels, *dt, *flux_eps, nlevels_nod2D_dev, area_inv_dev, fct_ttf_max_dev, fct_ttf_min_dev, fct_plus_dev, fct_min_dev);
    *alg_state = 6;
    transfer_back(*fct_plus, "fct_plus", alg_state);
    transfer_back(*fct_minus, "fct_minus", alg_state);
}

void fct_ale_inter_comm_acc_( int* alg_state, void**  fct_plus, void**  fct_minus, void**  fct_adf_v, int* myDim_nod2D, int* nl, void** nlevels_nod2D)
{
#if NUM_KERNELS < 7
    return;
#endif
    int maxLevels = *nl - 1;
    int* nlevels_nod2D_dev = reinterpret_cast<int*>(static_cast<gpuMemory*>(*nlevels_nod2D)->device_pointer);
    real_type* fct_adf_v_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_adf_v)->device_pointer);
    real_type* fct_plus_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_plus)->device_pointer);
    real_type* fct_min_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_minus)->device_pointer);
    fct_ale_b3_vertical<<< dim3(*myDim_nod2D), dim3(32) >>>(maxLevels, nlevels_nod2D_dev, fct_adf_v_dev, fct_plus_dev, fct_min_dev);
    transfer_back(*fct_adf_v, "fct_adf_v", alg_state);
    *alg_state = 7;
}

void fct_ale_post_comm_acc_( int* alg_state, void**  fct_plus, void**  fct_minus, void** fct_adf_h, int* myDim_edge2D, int* nl, void** nlevels_elem2D, int* nod_in_elem2D_dim, void** nod2D_edges, void** elem2D_edges)
{
#if NUM_KERNELS < 8
    return;
#endif
    int maxLevels = *nl - 1;
    int maxnElems = *nod_in_elem2D_dim;
    int* nlevels_elem2D_dev = reinterpret_cast<int*>(static_cast<gpuMemory*>(*nlevels_elem2D)->device_pointer);
    int* nod2D_edges_dev = reinterpret_cast<int*>(static_cast<gpuMemory*>(*nod2D_edges)->device_pointer);
    int* elem2D_edges_dev = reinterpret_cast<int*>(static_cast<gpuMemory*>(*elem2D_edges)->device_pointer);
    real_type* fct_adf_h_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_adf_h)->device_pointer);
    real_type* fct_plus_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_plus)->device_pointer);
    real_type* fct_min_dev = reinterpret_cast<real_type*>(static_cast<gpuMemory*>(*fct_minus)->device_pointer);

    fct_ale_b3_horizontal<<< dim3(*myDim_edge2D), dim3(32) >>>(maxLevels, nlevels_elem2D_dev, nod2D_edges_dev, elem2D_edges_dev, fct_adf_h_dev, fct_plus_dev, fct_min_dev);
    *alg_state = 9;
    transfer_back(*fct_adf_h, "fct_adf_h", alg_state);
}